#include "hip/hip_runtime.h"
/*
	BECHMARK BASICO PARA HARDWARE. POR CARLOS ANDRES GARNICA (HARDWARE ELITE)
	
	PROGRAMA QUE SUMA 2 VECTORES Y SU RESULTADO LO GUARDA EN UN NUEVO VECTOR.
	El programa realiza estas sumas tanto por CPU como por GPU.\nSe llenan de numero 
	aleatorio en arrays de 100 mil elementos, el numero aleatorio que salga se 
	guardara en ambos arrays a sumar, por lo que los resultados no varian ni para la 
	CPU ni para la GPU.

	NOTA: El programa solo sirve para PCs con GPU NVIDIA y que sean compatibles con la 
	tecnologia CUDA.
*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <random>
#include <limits>

#define ITER 100000

using namespace std;

typedef chrono::high_resolution_clock Clock;

// Version CPU de la funci�n suma de vectores
void vector_add_cpu(long long *a, long long *b, long long *c, long long n) {
	int i;
	for (i = 0; i < n; ++i) {
		c[i] = a[i] + b[i];
	}
}

// Versi�n GPU de la funci�n suma de vectores
__global__ void vector_add_gpu(long long *gpu_a, long long *gpu_b, long long *gpu_c, long long n) {
	int i = threadIdx.x;
	// No es necesario el loop for por que el runtime de CUDA
	// maneja estos hilos ITER (100.000) veces
	gpu_c[i] = gpu_a[i] + gpu_b[i];
	
}

// Funcion que obtiene un numero aleatorio para el llenado de los arrays
long long obtenerAleatorio() {
	random_device rd;
	mt19937 gen(rd());
	uniform_int_distribution<long long> dis(0, numeric_limits<int>::max());
	return dis(gen);
}

// Imprime el arreglo final sumado por el CPU
void imprimirArregloCPU(long long *c) {
	for (int i = 0; i < 15; i++) {
		cout << c[i] << " ";
	}
	cout << endl;
}

// Imprime el arreglo final sumado por el GPU
void imprimirArregloGPU(long long *gpu_c) {
	for (int i = 0; i < 15; i++) {
		cout << gpu_c[i] << " ";
	}
	cout << endl;
}

bool imprimirDatosGPUs() {
	int cantGPUs = 0;
	hipGetDeviceCount(&cantGPUs);
	cout << endl << "GPUs:\n______________________________" << endl;
	if (cantGPUs > 0) {
		cout << "Tarjetas Graficas nVIDIA detectadas: " << cantGPUs << "\nINFORMACION:\n" << endl;
	}
	else {
		cout << "No se detectaron GPUs nVIDIA. Recuerde tener los controladores de sus dispositivos actualizados o instalados." << endl;
		return false;
	}
	for (int i = 0; i < cantGPUs; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		cout << "Numero " << i;
		cout << endl << "Nombre: " << prop.name;
		cout << endl << "Reloj de Memoria (MHz): " << prop.memoryClockRate*0.001;
		cout << endl << "Ancho del bus de memoria (bits): " << prop.memoryBusWidth;
		cout << endl << "Ancho de banda maximo de memoria teorico (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6;
		cout << endl << "Reloj actual del GPU (MHz): " << prop.clockRate*0.001 << endl << endl;
	}
	return true;
}

int main() {

	long long *a, *b, *c;
	long long *gpu_a, *gpu_b, *gpu_c;

	a = (long long *)malloc(ITER * sizeof(long long));
	b = (long long *)malloc(ITER * sizeof(long long));
	c = (long long *)malloc(ITER * sizeof(long long));

	// Necesitamos variables accesibles en CUDA,
	// para eso hipMallocManaged nos las provee
	hipMallocManaged(&gpu_a, ITER * sizeof(long long));
	hipMallocManaged(&gpu_b, ITER * sizeof(long long));
	hipMallocManaged(&gpu_c, ITER * sizeof(long long));
	
	cout << "BECHMARK BASICO PARA HARDWARE. POR CARLOS ANDRES GARNICA (HARDWARE ELITE)" << endl << endl;
	cout << "PROGRAMA QUE SUMA 2 VECTORES Y SU RESULTADO LO GUARDA EN UN NUEVO VECTOR." << endl;
	cout << "El programa realiza estas sumas tanto por CPU como por GPU.\nSe llenan de numero aleatorio en arrays de 100 mil elementos, el numero aleatorio que salga se guardara en ambos arrays a sumar, por lo que los resultados no varian ni para la CPU ni para la GPU" << endl;
	//SI NO SE DETECTAN DISPOSITIVOS NVIDIA EN EL PC, SE TERMINA LA EJECUCION
	if (!imprimirDatosGPUs()) {
		return 0;
	}
	// SE LLENA LAS MATRICAS A Y B TANTO DE LA GPU COMO DE LA CPU PARA HACER LAS SUMAS ALEATORIAS
	for (int i = 0; i < ITER; ++i) {
		long long numAleatorio = obtenerAleatorio();
		a[i] = numAleatorio;
		b[i] = numAleatorio;
		
		gpu_a[i] = numAleatorio;
		gpu_b[i] = numAleatorio;
	}
	cout << "______________________________\n";
	cout << "TIEMPOS: " << endl;
	// Llama a la versi�n CPU y la temporiza
	auto cpu_start = Clock::now();
	vector_add_cpu(a, b, c, ITER);
	auto cpu_end = Clock::now();
	cout << "Suma de vectores con la CPU: "<< chrono::duration_cast<chrono::nanoseconds>(cpu_end - cpu_start).count()<< " nanosegundos.\n";

	// Llama a la versi�n GPU y la temporiza
	// Los triples <> es una extensi�n del runtime CUDA que permite
	// que los parametros de una llamada al kernel CUDA sean pasados
	// En este ejemplo estamos pasando un thread block con ITER threads
	auto gpu_start = Clock::now();
	vector_add_gpu <<<1, ITER >>> (gpu_a, gpu_b, gpu_c, ITER);
	hipDeviceSynchronize();
	auto gpu_end = Clock::now();
	cout << "Suma de vectores con la GPU: "<< chrono::duration_cast<chrono::nanoseconds>(gpu_end - gpu_start).count() << " nanosegundos.\n";

	/*
		IMPRIME LOS RESULTADOS DE LAS SUMAS GUARDADAS EN LOS VECTORES CORRESPONDIENTES, TANTO DEL CPU COMO DEL GPU. SI AMBOS SON IGUALES, ES CORRECTO LOS
		DATOS DE TIEMPO DEL BECHMARCK
	*/
	cout << "______________________________\n";
	cout << "Resultados de las sumas (Se imprimen los primeros 15 resultados del arreglo): " << endl;
	cout << "CPU: " << endl;
	imprimirArregloCPU(c);
	cout << "GPU: " << endl;
	imprimirArregloGPU(c);

	//LIBERAR MEMORIA DE LA GRAFICA
	hipFree(a);
	hipFree(b);
	hipFree(c);

	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);

	// Libere la memoria basada en la funci�n CPU
	free(a);
	free(b);
	free(c);
	system("pause");
	return 0;
}
